#include "hip/hip_runtime.h"
/*************************************************************************************/
/*      Copyright 2009 Barcelona Supercomputing Center                               */
/*                                                                                   */
/*      This file is part of the NANOS++ library.                                    */
/*                                                                                   */
/*      NANOS++ is free software: you can redistribute it and/or modify              */
/*      it under the terms of the GNU Lesser General Public License as published by  */
/*      the Free Software Foundation, either version 3 of the License, or            */
/*      (at your option) any later version.                                          */
/*                                                                                   */
/*      NANOS++ is distributed in the hope that it will be useful,                   */
/*      but WITHOUT ANY WARRANTY; without even the implied warranty of               */
/*      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                */
/*      GNU Lesser General Public License for more details.                          */
/*                                                                                   */
/*      You should have received a copy of the GNU Lesser General Public License     */
/*      along with NANOS++.  If not, see <http://www.gnu.org/licenses/>.             */
/*************************************************************************************/

/*
<testinfo>
test_generator=gens/gpu-generator
test_CXX=nvcc
</testinfo>
*/

#define __aligned__ ignored
#include "gpuprocessor.hpp"
#include "nanos.h"
#undef __aligned__
#include <iostream>
#include <stdlib.h>
#include <string.h>


using namespace std;

using namespace nanos;
using namespace nanos::ext;

#define NTASKS 32
const int ntasks = NTASKS;

#define N 512
const int n = 512;

// Arguments' struct
typedef struct {
   int * err;
   int n;
   int idx;
   int * a;
   int * b;
   int * c;
} test_args;


// GPU kernels
__global__ void check_task_params ( int idx, int * a , int * b, int * c, int * err );


// GPU tasks
///#pragma omp target device (cuda) copydeps
///#pragma omp task input(n, idx, a) output(b, err) inout(c)
//void test_gpu_task ( int * err, int n, int idx, int * a, int * b, int * c );
void test_gpu_task ( void * args );


///#pragma omp target device (cuda) copydeps
///#pragma omp task 
void dummy_task ( void * args );


int main ( int argc, char **argv )
{
   std::cout << "Testing GPU task parameter copy mechanism" << std::endl;

   int i, j;
   test_args ** args = new test_args*[ntasks];

   nanos::WG *wg = nanos::getMyThreadSafe()->getCurrentWD();

   for ( i = 0; i < ntasks; i++ ) {
      // Data structures initialization
      args[i] = new test_args();
      args[i]->err = new int(1);
      args[i]->n = n;
      args[i]->idx = i;

      args[i]->a = new int[n];
      args[i]->b = new int[n];
      args[i]->c = new int[n];

      args[i]->a[0] = i;
      args[i]->b[0] = 0;
      args[i]->c[0] = i;

      for ( j = 1; j < n; j++ ) {
         args[i]->a[j] = args[i]->a[j-1] + 1;
         args[i]->b[j] = 0;
         args[i]->c[j] = args[i]->c[j-1] + 1;
      }

      // Parameter copies
      CopyData* cd = new CopyData[6];
      // CopyData( address, attribute, input, output, size )
      cd[0] = CopyData( (uint64_t)args[i]->err, NANOS_SHARED, true, true, sizeof(int) );
      cd[1] = CopyData( (uint64_t)&args[i]->n, NANOS_PRIVATE, true, false, sizeof(args[i]->n) );
      cd[2] = CopyData( (uint64_t)&args[i]->idx, NANOS_PRIVATE, true, false, sizeof(args[i]->idx) );
      cd[3] = CopyData( (uint64_t)args[i]->a, NANOS_SHARED, true, false, n * sizeof(int) ); 
      cd[4] = CopyData( (uint64_t)args[i]->b, NANOS_SHARED, false, true, n * sizeof(int) ); 
      cd[5] = CopyData( (uint64_t)args[i]->c, NANOS_SHARED, true, true, n * sizeof(int) );

      // Task execution
      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_gpu_task ), sizeof( args[i] ), args[i], 6, cd );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
   }

   usleep(500);

   wg->waitCompletion();

   // waitCompletion does not wait for the last copy-out to finish when overlapping data and computation,
   // so create several dummy tasks in order to force last data to be copied. (See #307)
   for ( i = 0; i < ntasks; i++ ) {
      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( dummy_task ), 0, 0, 0, 0 );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
   }

   usleep(500);

   wg->waitCompletion();

   for ( i = 0; i < ntasks; i++ ) {
      if ( *args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << *args[i]->err << " errors found: task parameter copy mechanism did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   delete[] args;

   std::cout << "   Execution ... ok!" << std::endl;
   
   std::cout << "End testing GPU task parameter copy mechanism" << std::endl;
    
   
   return 0;
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task 
void dummy_task ( void * args )
{
   int a = 4985;
   float b = 0.6234;
   double c = 0.0;

   int i;
   for ( i = 0; i < NTASKS*NTASKS*NTASKS*NTASKS; i++ ) {
      c += (double) (a * b) - (b / a) + (a + b);
   }
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task input(n, idx, a) output(b, err) inout(c)
void test_gpu_task ( void * args )
{

   test_args targs;

   nanos_get_addr(0, ( void ** ) &targs.err);
   targs.n = ( (test_args *) args)->n;
   targs.idx = ( (test_args *) args)->idx;
   nanos_get_addr(3, ( void ** ) &targs.a);
   nanos_get_addr(4, ( void ** ) &targs.b);
   nanos_get_addr(5, ( void ** ) &targs.c);

   int i, err, value;

   size_t size = targs.n * sizeof ( int );
   int err_h[targs.n];
   int * err_d;
   hipMalloc( &err_d, size );

   memset( err_h, 0x1, size );
   hipMemcpy( err_d, err_h, size, hipMemcpyHostToDevice );

   check_task_params <<< 1, targs.n >>> ( targs.idx, targs.a, targs.b, targs.c, err_d );

   hipDeviceSynchronize();

   hipMemcpy( err_h, err_d, size, hipMemcpyDeviceToHost );

   err = 0;
   for ( i = 0; i < targs.n; i++ ) {
      // Error checking for A and C, from kernel execution (as inputs)
      if ( err_h[i] ) {
         std::cout << "[" << targs.idx << "] Input error detected at position " << i << ": " << err_h[i] << std::endl;
         err++;
      }
   }

   // Error checking for B, from kernel execution (as output)
   hipMemcpy( err_h, targs.b, size, hipMemcpyDeviceToHost );
   value = targs.idx;
   for ( i = 0; i < targs.n; i++ ) {
      if ( err_h[i] != value ) {
         std::cout << "[" << targs.idx << "] Output error detected at position " << i << ": " << err_h[i] << std::endl;
         err++;
      }
      value++;
   }

   // Write the result through hipMemcpy, as 'targs->err' address belongs to GPU memory space
   hipMemcpy( targs.err, &err, sizeof( int ), hipMemcpyHostToDevice );

   hipFree(err_d);

}


/***** GPU CODE *****/
__global__ void check_task_params ( int idx, int * a , int * b, int * c, int * err )
{
   int i = threadIdx.x;
   int val = idx + threadIdx.x;

   err[i] = 0;

   err[i] += a[i] - val;
   err[i] += c[i] - val;

   b[i] = val;

}
/***** END GPU CODE *****/

