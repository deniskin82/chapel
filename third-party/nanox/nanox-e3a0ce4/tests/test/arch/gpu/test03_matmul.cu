#include "hip/hip_runtime.h"
/*************************************************************************************/
/*      Copyright 2009 Barcelona Supercomputing Center                               */
/*                                                                                   */
/*      This file is part of the NANOS++ library.                                    */
/*                                                                                   */
/*      NANOS++ is free software: you can redistribute it and/or modify              */
/*      it under the terms of the GNU Lesser General Public License as published by  */
/*      the Free Software Foundation, either version 3 of the License, or            */
/*      (at your option) any later version.                                          */
/*                                                                                   */
/*      NANOS++ is distributed in the hope that it will be useful,                   */
/*      but WITHOUT ANY WARRANTY; without even the implied warranty of               */
/*      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                */
/*      GNU Lesser General Public License for more details.                          */
/*                                                                                   */
/*      You should have received a copy of the GNU Lesser General Public License     */
/*      along with NANOS++.  If not, see <http://www.gnu.org/licenses/>.             */
/*************************************************************************************/

/*
<testinfo>
test_generator=gens/gpu-generator
test_CXX=nvcc
</testinfo>
*/

#define __aligned__ ignored
#include "gpuprocessor.hpp"
#include "nanos.h"
#undef __aligned__
#include <iostream>
#include <stdlib.h>
#include <string.h>


using namespace std;

using namespace nanos;
using namespace nanos::ext;

// Block size
#define BSIZE 8
const int bsize = BSIZE;

// Number of blocks
#define NB 4
const int nb = NB;

// Arguments' struct
typedef struct {
   int bs;
   float * a;
   float * b;
   float * c;
} test_args;


// Auxiliar functions
void matmul_host ( int bs, float * a, float * b, float * c );


// GPU kernel
__global__ void matmul_gpu ( float * a , float * b, float * c );


// GPU task
///#pragma omp target device (cuda) copydeps
///#pragma omp task input(a, b) inout(c)
//void matmul_task ( int nb, int bs, float * a, float * b, float * c );
void matmul_task ( void * args );


nanos_smp_args_t test_device_arg = { matmul_task };

int main ( int argc, char **argv )
{
   std::cout << "Testing matrix multiply on GPU" << std::endl;

   int i, j, k, n;
   n = nb * bsize;

   // Data structures initialization
   float * a = new float[n * n];
   float * b = new float[n * n];
   float * c = new float[n * n];

   for ( i = 0; i < n; i++ ) {
      for ( j = 0; j < n; j++ ) {
         a[i * n + j] = 1.0;
         b[i * n + j] = 2.0;
         c[i * n + j] = 0.0;
      }
   }

   // Task creation
   nanos::WG *wg = nanos::getMyThreadSafe()->getCurrentWD();

   for ( i = 0; i < nb; i++ ) {
      for ( j = 0; j < nb; j++ ) {
         for ( k = 0; k < nb; k++ ) {
            nanos_wd_t wd = 0;
            
            test_args *args = 0;
            
            
            //nanos_device_t test_devices[1] = { {NANOS_GPU_DESC( test_device_arg )} };
            nanos_device_t test_devices[] = {
               {
                  nanos_gpu_factory,
                  nanos_gpu_dd_size,
                  &test_device_arg

               }
            };
            
            nanos_wd_props_t props = {
               .mandatory_creation = true,
            };
            props.tied = 1;
            
            nanos_copy_data_t *cd = 0;

//            nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( matmul_task ), sizeof( args ), &args, 4, cd );
//            wd->tied();
//            wg->addWork( *wd );
//            nanos::sys.submitWithDependencies( *wd, 1, dep );
            
            NANOS_SAFE( nanos_create_wd ( &wd, 1, test_devices , sizeof( test_args ), (void **) &args, nanos_current_wd(), &props, 4, &cd ) );
            
            args->bs = bsize;
            args->a = &a[i * nb * bsize * bsize + k * bsize * bsize];
            args->b = &b[k * nb * bsize * bsize + j * bsize * bsize];
            args->c = &c[i * nb * bsize * bsize + j * bsize * bsize];
            
            cd[0].address = (uint64_t) &args->bs;
            cd[0].sharing = NANOS_PRIVATE;
            cd[0].flags.input = true;
            cd[0].flags.output = false;
            cd[0].size = sizeof(int);
            //cd[0] = (nanos_copy_data_t) { (uint64_t) &args->bs, NANOS_PRIVATE, { true, false }, sizeof(int) };
            
            cd[1].address = (uint64_t) args->a;
            cd[1].sharing = NANOS_SHARED;
            cd[1].flags.input = true;
            cd[1].flags.output = false;
            cd[1].size = bsize * bsize * sizeof(float);
            //cd[1] = (nanos_copy_data_t) { (uint64_t) args->a, NANOS_SHARED, { true, false }, bsize * bsize * sizeof(float) };
            
            cd[2].address = (uint64_t) args->b;
            cd[2].sharing = NANOS_SHARED;
            cd[2].flags.input = true;
            cd[2].flags.output = false;
            cd[2].size = bsize * bsize * sizeof(float);
            //cd[2] = (nanos_copy_data_t) { (uint64_t) args->b, NANOS_SHARED, { true, false }, bsize * bsize * sizeof(float) };
            
            cd[3].address = (uint64_t) args->c;
            cd[3].sharing = NANOS_SHARED;
            cd[3].flags.input = true;
            cd[3].flags.output = true;
            cd[3].size = bsize * bsize * sizeof(float);
            //cd[3] = (nanos_copy_data_t) { (uint64_t) args->c, NANOS_SHARED, { true, true }, bsize * bsize * sizeof(float) };

            nanos_dependence_t deps[3] = {
               {
                  (void **) args->a,
                  ((char *) (&a[i * nb * bsize * bsize + k * bsize * bsize]) - (char *) args->a),
                  { 1, 0, 0 },
                  sizeof(float) * bsize * bsize
               },
               {
                  (void **) args->b,
                  ((char *) (&b[k * nb * bsize * bsize + j * bsize * bsize]) - (char *) args->b),
                  { 1, 0, 0 },
                  sizeof(float) * bsize * bsize
               },
               {
                  (void **) args->c,
                  ((char *) (&c[i * nb * bsize * bsize + j * bsize * bsize]) - (char *) args->c),
                  { 1, 1, 0 },
                  sizeof(float) * bsize * bsize
               }
            };
            
            NANOS_SAFE( nanos_submit( wd, 1, deps, 0 ) );
            
            usleep(500);
         }
      }
   }


   NANOS_SAFE( nanos_wg_wait_completion( nanos_current_wd() ) );

   // Error checking
   int err = 0;
   for ( i = 0; i < n; i++ ) {
      for ( j = 0; j < n; j++ ) {
         if ( c[i*n+j] != bsize * nb * 2 ) {
        	 std::cout << "Error at " << i << ", " << j << ": (GPU) " << c[i*n+j] << " vs (CPU) " << bsize * nb * 2 << std::endl;
            err++;
         }
      }
   }

   if ( err ) {
         std::cout << err << " errors found: matrix multiply did not succeed.   FAIL" << std::endl;
         exit( 1 );
   }
   
   std::cout << "   Execution ... ok!" << std::endl;
   
   std::cout << "End testing matrix multiply on GPU" << std::endl;
    
   
   return 0;
}



///#pragma omp target device (cuda) copydeps
///#pragma omp task input(n, idx, a) output(b, err) inout(c)
void matmul_task ( void * args )
{

   test_args * targs = new test_args();

   targs->bs = ( (test_args *) args)->bs;
   nanos_get_addr(1, ( void ** ) &targs->a);
   nanos_get_addr(2, ( void ** ) &targs->b);
   nanos_get_addr(3, ( void ** ) &targs->c);
   
   dim3 dimBlock(targs->bs, targs->bs);
   matmul_gpu <<< 1, dimBlock >>> ( targs->a, targs->b, targs->c );
   
   
   hipDeviceSynchronize();
   
   delete targs;

}


/***** GPU CODE *****/
__global__ void matmul_gpu ( float * a , float * b, float * c )
{
   int idx = threadIdx.x * blockDim.x + threadIdx.y;
   int i = threadIdx.x;
   int j = threadIdx.y;

   int k;
   for ( k = 0; k < blockDim.x; k++ ) {
      c[idx] += a[i*blockDim.x + k] * b[k*blockDim.x + j];
   }
}
/***** END GPU CODE *****/

