#include "hip/hip_runtime.h"
/*************************************************************************************/
/*      Copyright 2009 Barcelona Supercomputing Center                               */
/*                                                                                   */
/*      This file is part of the NANOS++ library.                                    */
/*                                                                                   */
/*      NANOS++ is free software: you can redistribute it and/or modify              */
/*      it under the terms of the GNU Lesser General Public License as published by  */
/*      the Free Software Foundation, either version 3 of the License, or            */
/*      (at your option) any later version.                                          */
/*                                                                                   */
/*      NANOS++ is distributed in the hope that it will be useful,                   */
/*      but WITHOUT ANY WARRANTY; without even the implied warranty of               */
/*      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                */
/*      GNU Lesser General Public License for more details.                          */
/*                                                                                   */
/*      You should have received a copy of the GNU Lesser General Public License     */
/*      along with NANOS++.  If not, see <http://www.gnu.org/licenses/>.             */
/*************************************************************************************/

/*
<testinfo>
test_generator=gens/gpu-generator
test_CXX=nvcc
</testinfo>
*/

#define __aligned__ ignored
#include "gpuprocessor.hpp"
#undef __aligned__
#include <iostream>
#include <stdlib.h>
#include <string.h>


using namespace std;

using namespace nanos;
using namespace nanos::ext;

#define NREP 100

// Arguments' struct
typedef struct {
   int err;
   int n;
   int * Ah;
   int * Ad;
   int * Bh;
   int * Bd;
   int task;
} test_args;


// GPU kernels
__global__ void check_host_to_device ( int * a , int * err );
__global__ void check_device_to_device ( int * a, int * err );


// GPU tasks
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_init ( void * args );
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_host_to_device ( void * args );
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_device_to_device ( void * args );
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_device_to_host ( void * args );
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_cleanup ( void * args );


int main ( int argc, char **argv )
{
   std::cout << "Testing GPU memory manager ( GPUDevice ): allocate, free, copy in / out / local" << std::endl;

   int i, n = 512;
   int ngpus = nanos::ext::GPUDD::getGPUCount();
   test_args ** args = new test_args* [ngpus];
   
   for ( i = 0; i < ngpus; i++ ) {
      args[i] = new test_args();
      args[i]->err = 0;
      args[i]->n = n;
      args[i]->Ah = 0;
      args[i]->Ad = 0;
      args[i]->Bh = 0;
      args[i]->Bd = 0;
      args[i]->task = 0;
   }

   nanos::WG *wg = nanos::getMyThreadSafe()->getCurrentWD();

   // Execution of tasks: for every kind of task, create 100 times the number of GPUs, so that we
   // assume that each GPU will pick at least one task of each type.
   // 'task' member controls that each kind of task is executed once and only once on the same GPU.

   // Initialization
   for ( i = 0; i < ngpus*NREP; i++ ) {
///      test_init( args[i] );

      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_init ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
      
      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: data allocation into device memory did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Initialization ... ok!" << std::endl;
   
   // First copy: host to device
   for ( i = 0; i < ngpus*NREP; i++ ) {
///      test_host_to_device( args[i] );

      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_host_to_device ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
      
      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: copying memory from host to device did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Host --> Device ... ok!" << std::endl;
   
   // Second copy: device to device (local)
   for ( i = 0; i < ngpus*NREP; i++ ) {
///      test_device_to_device( args[i] );
      
      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_device_to_device ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
      
      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: copying memory from device to device did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Device --> Device ... ok!" << std::endl;

   // Third copy: device to host
   for ( i = 0; i < ngpus*NREP; i++ ) {
///      test_device_to_host( args[i] );
      
      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_device_to_host ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );

      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: copying memory from device to host did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Device --> Host ... ok!" << std::endl;
   
   // CLEANUP
   for ( i = 0; i < ngpus*NREP; i++ ) {
///      test_cleanup( args[i] );

      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_cleanup ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
      
      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: data cleanup from device memory did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Clean up ... ok!" << std::endl;
  
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->task != 5 ) {
         std::cout << "WARNING: thread " << i << " did not run all kinds of tasks: " << args[i]->task << std::endl;
      }
   }
 
   delete args;

   std::cout << "End testing GPU memory manager ( GPUDevice )" << std::endl;
    
   
   return 0;
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_init ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::getMyThreadSafe())->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];

   if ( targs->task != 0 )
      return;

   targs->task++;

   size_t size = targs->n * sizeof ( int );
   targs->Ah = new int[targs->n];
   targs->Bh = new int[targs->n];
   targs->Ad = ( int * ) GPUDevice::allocate( size );
   targs->Bd = ( int * ) GPUDevice::allocate( size );

   targs->err = 0;

   if ( targs->Ah == 0 ) targs->err++;
   if ( targs->Bh == 0 ) targs->err++;
   if ( targs->Ad == 0 ) targs->err++;
   if ( targs->Bd == 0 ) targs->err++;
   
   usleep( 50 );
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_cleanup ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::getMyThreadSafe())->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   if ( targs->task != 4 )
      return;
   
   targs->task++;

   delete targs->Ah;
   delete targs->Bh;
   GPUDevice::free( targs->Ad );
   GPUDevice::free( targs->Bd );
   
   targs->err = 0;
   
   usleep( 50 );
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_host_to_device ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::getMyThreadSafe())->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   if ( targs->task != 1 )
      return;
   
   targs->task++;

   hipStream_t inStream = ((nanos::ext::GPUProcessor *) getMyThreadSafe()->runningOn())->getGPUProcessorInfo()->getInTransferStream();
   
   int i;   
   size_t size = targs->n * sizeof ( int );

   // Initalize arrays
   for ( i = 0; i < targs->n; i++ ) {
      targs->Ah[i] = i;
      targs->Bh[i] = 1;
   }
   hipMemset( targs->Ad, 0, size );
   hipMemset( targs->Bd, 0, size );

   GPUDevice::copyIn( targs->Ad, ( uint64_t ) targs->Ah, size );
   
   hipStreamSynchronize( inStream );
   
   // Launch a kernel to check the copy was successful and get the result back from the GPU
   // Arrays Bd and Bh will contain the error checking result
   check_host_to_device <<< 1, targs->n >>> ( targs->Ad, targs->Bd );
   hipMemcpy( targs->Bh, targs->Bd, size, hipMemcpyDeviceToHost );

   targs->err = 0;
   for ( i = 0; i < targs->n; i++ ) {
      if ( targs->Bh[i] ) {
         std::cout << "Error detected at position " << i << ": " << targs->Bh[i] << std::endl;
         targs->err++;
      }
   }
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_device_to_device ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::getMyThreadSafe())->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   if ( targs->task != 2 )
      return;
   
   targs->task++;

   hipStream_t inStream = ((nanos::ext::GPUProcessor *) getMyThreadSafe()->runningOn())->getGPUProcessorInfo()->getInTransferStream();
   
   int i;   
   size_t size = targs->n * sizeof ( int );
   
   // Initalize arrays
   for ( i = 0; i < targs->n; i++ ) {
      targs->Ah[i] = targs->n - i;
      targs->Bh[i] = 1;
   }
   hipMemcpy( targs->Ad, targs->Ah, size, hipMemcpyHostToDevice );
   hipMemset( targs->Bd, 0, size );

   GPUDevice::copyLocal( targs->Bd, targs->Ad, size );
   
   hipStreamSynchronize( inStream );
   
   // Launch a kernel to check the copy was successful and get the result back from the GPU
   // Arrays Ad and Bh will contain the error checking result
   check_device_to_device <<< 1, targs->n >>> ( targs->Bd, targs->Ad );
   hipMemcpy( targs->Bh, targs->Ad, size, hipMemcpyDeviceToHost );

   targs->err = 0;
   for ( i = 0; i < targs->n; i++ ) {
      if ( targs->Bh[i] ) {
         std::cout << "Error detected at position " << i << ": " << targs->Bh[i] << std::endl;
         targs->err++;
      }
   }
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_device_to_host ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::getMyThreadSafe())->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   if ( targs->task != 3 )
      return;
   
   targs->task++;

   int i;   
   size_t size = targs->n * sizeof ( int );
   
   // Initialize arrays
   for ( i = 0; i < targs->n; i++ ) {
      targs->Ah[i] = i;
   }
   memset( targs->Bh, 0, size ); 
   hipMemcpy( targs->Bd, targs->Ah, size, hipMemcpyHostToDevice );

   // As asynchronous copies involve the cache management, make always synchronous copy out's
   GPUDevice::copyOutSyncToHost( targs->Bh, targs->Bd, size );

   ( ( GPUThread * ) getMyThreadSafe())->executePendingCopies();

   targs->err = 0;
   for ( i = 0; i < targs->n; i++ ) {
      if ( targs->Bh[i] != targs->Ah[i] ) {
         std::cout << "Error detected at position " << i << ": " << targs->Bh[i] << std::endl;
         targs->err++;
      }
   }
}



/***** GPU CODE *****/
__global__ void check_host_to_device ( int * a , int * err )
{
   int i = threadIdx.x;

   err[i] = a[i] - i;

   a[i] = blockDim.x - threadIdx.x;

}

__global__ void check_device_to_device ( int * b, int * err )
{
   int i = threadIdx.x;

   err[i] = b[i] - (blockDim.x - i);

   b[i] = threadIdx.x;

}
/***** END GPU CODE *****/

